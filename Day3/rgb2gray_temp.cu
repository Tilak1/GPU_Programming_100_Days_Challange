#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "timer.h"

__global__ void rgb2gray_kernel(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        gray[idx] = (unsigned char)(0.299f * red[idx] + 0.587f * green[idx] + 0.114f * blue[idx]);
    }
}

void rgb2gray_gpu(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height) {
    Timer timer;

    // Allocate GPU memory
    startTime(&timer);
    unsigned char *red_d, *green_d, *blue_d, *gray_d;
    hipMalloc((void**)&red_d, width * height * sizeof(unsigned char));
    hipMalloc((void**)&green_d, width * height * sizeof(unsigned char));
    hipMalloc((void**)&blue_d, width * height * sizeof(unsigned char));
    hipMalloc((void**)&gray_d, width * height * sizeof(unsigned char));
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(red_d, red, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(green_d, green, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    startTime(&timer);
    rgb2gray_kernel<<<gridSize, blockSize>>>(red_d, green_d, blue_d, gray_d, width, height);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel execution time");

    // Copy result back to host
    startTime(&timer);
    hipMemcpy(gray, gray_d, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy back to host time");

    // Free memory
    hipFree(red_d);
    hipFree(green_d);
    hipFree(blue_d);
    hipFree(gray_d);
}
